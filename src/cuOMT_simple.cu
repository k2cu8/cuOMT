#include "hip/hip_runtime.h"
#include "cuOMT_simple.cuh"
#include <hiprand.h>
#include <iomanip>



int cuOMT_simple::GPU_generate_RNM(float* P, const int nRowP, const int nColP)
{
	int seed = (int)clock();
	thrust::device_ptr<float> P_ptr(P);
	thrust::transform(
		thrust::make_counting_iterator(seed),
		thrust::make_counting_iterator(seed + nRowP * nColP),
		P_ptr,
		GenRandFloat());
	return 0;
}

int cuOMT_simple::GPU_generate_RNM(float* P, const int nRowP, const int nColP, float lower_b, float upper_b)
{
	int seed = (int)clock();
	thrust::device_ptr<float> P_ptr(P);
	thrust::transform(
		thrust::make_counting_iterator(seed),
		thrust::make_counting_iterator(seed + nRowP * nColP),
		P_ptr,
		GenRandFloatUni(lower_b, upper_b));
	return 0;
}

int cuOMT_simple::curand_RNG_sobol(float* P, const int nRowP, const int nColP, unsigned long long offset)
{
    using namespace std;
    hiprandStatus_t curandResult;
    hiprandGenerator_t qrng;

    curandResult = hiprandCreateGenerator(&qrng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not create quasi-random number generator: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    curandResult = hiprandSetQuasiRandomGeneratorDimensions(qrng, nColP);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not set number of dimensions for quasi-random number generator: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    curandResult = hiprandSetGeneratorOrdering(qrng, HIPRAND_ORDERING_QUASI_DEFAULT);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not set order for quasi-random number generator: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    curandResult = hiprandSetGeneratorOffset(qrng, offset);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not set offset for quasi-random number generator: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    curandResult = hiprandGenerateUniform(qrng, P, nRowP * nColP);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not generate quasi-random numbers: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    curandResult = hiprandDestroyGenerator(qrng);
    if (curandResult != HIPRAND_STATUS_SUCCESS)
    {
        string msg("Could not destroy quasi-random number generator: ");
        msg += curandResult;
        throw std::runtime_error(msg);
    }

    thrust::device_ptr<float> P_ptr(P);
    thrust::transform(P_ptr, P_ptr + nRowP * nColP, P_ptr, axpb<float>(1, 0.5));

    return 0;
}

int cuOMT_simple::gd_init(int argc, char* argv[])
{
	/*set problem parameters*/
	d_eps.resize(1);
	d_eps[0] = eps;

	/*allocate device vectors*/
	d_P = 0;
	d_volP = 0;
	
	d_U = 0;
	d_PX = 0;
#ifdef USE_FANCY_GD
    d_adam_m = 0;
    d_adam_v = 0;
#endif

	// printf("cuOMT running...\n");

	if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! CUBLAS initialization error\n");
		return EXIT_FAILURE;
	}

	/* allocate device vectors*/
	if (hipMalloc((void **)&d_P, numP * dim * sizeof(d_P[0])) != hipSuccess)
	{
		fprintf(stderr, "!!!! device memory allocation error (allocate P)\n");
		return EXIT_FAILURE;
	}

	if (hipMalloc((void **)&d_volP, voln * dim * sizeof(d_volP[0])) != hipSuccess)
	{
		fprintf(stderr, "!!!! device memory allocation error (allocate volP)\n");
		return EXIT_FAILURE;
	}
#ifdef USE_FANCY_GD
    if (hipMalloc((void **)&d_adam_m, numP * sizeof(d_adam_m[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate d_adam_m)\n");
        return EXIT_FAILURE;
    }

    if (hipMalloc((void **)&d_adam_v, numP * sizeof(d_adam_v[0])) != hipSuccess)
    {
        fprintf(stderr, "!!!! device memory allocation error (allocate d_adam_v)\n");
        return EXIT_FAILURE;
    }
#endif
	if (hipMalloc((void **)&d_U, numP * voln * sizeof(d_U[0])) != hipSuccess)
	{
		fprintf(stderr, "!!!! device memory allocation error (U)\n");
		return EXIT_FAILURE;
	}


	/*fill in data*/

	// set parameters from command line arguments: P, A, h, adam_m, adam_v
	d_P_ptr = thrust::device_pointer_cast(d_P);
	d_A.resize(numP);
	d_h.resize(numP);
	bool set_P(false), set_A(false), set_h(false);
#ifdef USE_FANCY_GD
    d_adam_m_ptr = thrust::device_pointer_cast(d_adam_m);
    d_adam_v_ptr = thrust::device_pointer_cast(d_adam_v);
    bool set_adam_m(false), set_adam_v(false);
#endif
	for (int i = 1; i < argc; ++i)
	{
		if (strcmp(argv[i], "-P") == 0)
		{
			_set_from_csv<float>(argv[i + 1], d_P, numP, dim);
			set_P = true;

            std::cout << "Read P successfully." << std::endl;	
		}

		if (strcmp(argv[i], "-A") == 0)
		{
			_set_from_csv<float>(argv[i + 1], thrust::raw_pointer_cast(&d_A[0]), 1, numP);
			set_A = true;

            std::cout << "Read target measure successfully." << std::endl;
		}

		if (strcmp(argv[i], "-h") == 0)
		{
			_set_from_csv<float>(argv[i + 1], thrust::raw_pointer_cast(&d_h[0]), 1, numP);
			set_h = true; 

            std::cout << "Read initial h successfully." << std::endl;
		}
#ifdef USE_FANCY_GD
        if (strcmp(argv[i], "-adam_m") == 0)
        {
            _set_from_csv<float>(argv[i + 1], d_adam_m, 1, numP);
            set_adam_m = true;


            std::cout << "Read initial adam_m successfully." << std::endl;
        }

        if (strcmp(argv[i], "-adam_v") == 0)
        {
            _set_from_csv<float>(argv[i + 1], d_adam_v, 1, numP);
            set_adam_v = true;


            std::cout << "Read initial adam_v successfully." << std::endl;
        }
#endif
	}
	if (!set_P)
	{
		/*_set_random_parameter(d_P, numP, dim);
		thrust::transform(d_P_ptr, d_P_ptr + numP * dim, d_P_ptr, axpb<float>(1, -0.5));*/
		GPU_generate_RNM(d_P, numP, dim, -0.5f, 0.5f);

	}
	if (!set_A)
	{
		thrust::fill(d_A.begin(), d_A.end(), 1.0f / numP);
	}
	if (!set_h)
	{
		thrust::fill(d_h.begin(), d_h.end(), 0.0f);
	}
#ifdef USE_FANCY_GD
    if (!set_adam_m)
    {
        thrust::fill(d_adam_m_ptr, d_adam_m_ptr + numP, 0.0f);
    }
    if (!set_adam_v)
    {
        thrust::fill(d_adam_v_ptr, d_adam_v_ptr + numP, 0.0f);
    }
#endif
	// volP
	d_volP_ptr = thrust::device_pointer_cast(d_volP);

	// ind
	d_ind.resize(voln);
	thrust::fill(d_ind.begin(), d_ind.end(), -1);
	d_ind_val.resize(voln);
	thrust::fill(d_ind_val.begin(), d_ind_val.end(), 0.0f);

	// g
	d_g.resize(numP);



	/*---gradient descent---*/
	iter = 0;
	d_g_norm.resize(1);

	d_U_ptr = thrust::device_pointer_cast(d_U);

	d_delta_h.resize(numP);

	// loop parameters
	d_voln_key.resize(voln);

	// sample_id
	d_sampleId.resize(numP*voln);
	thrust::host_vector<int> h_sampleId(numP * voln);
	for (int i = 0; i < numP; ++i)
		for (int j = 0; j < voln; ++j)
		{
			h_sampleId[i + j * numP] = j;
		}
	thrust::copy(h_sampleId.begin(), h_sampleId.end(), d_sampleId.begin());
	h_sampleId.clear();


	// cell_id
	d_cellId.resize(numP*voln);
	thrust::host_vector<int> h_cellId(numP * voln);
	for (int i = 0; i < numP; ++i)
		for (int j = 0; j < voln; ++j)
		{
			h_cellId[i + j * numP] = i;
		}
	thrust::copy(h_cellId.begin(), h_cellId.end(), d_cellId.begin());
	h_cellId.clear();


	// duplicate of histogram by dimension
	d_Hist.resize(numP * dim);

	// dummy key for reduction in calculating knn
	d_knn_dummy_key.resize(numP);

	// voln * numP stride used to replicate vector dim times
	d_numP_voln_rep_ptr = thrust::device_pointer_cast(thrust::raw_pointer_cast(&d_cellId[0]));

	tic();

	return 0;
}

int cuOMT_simple::gd_pre_calc()
{
	//tic();

	// fill volP with random numbers
	
	GPU_generate_RNM(d_volP, voln, dim, -.5f, .5f);

	return 0;
}

int cuOMT_simple::gd_calc_measure()
{
	// duplicate h, i.e repmat(h, [voln 1])
	thrust::gather(d_numP_voln_rep_ptr, d_numP_voln_rep_ptr + numP*voln, d_h.begin(), d_U_ptr);


	// PX+H
	//thrust::transform(d_U_ptr, d_U_ptr + numP * voln, d_PX_ptr, d_U_ptr, thrust::plus<float>());
	if (hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, numP, voln, dim, alpha, d_P, numP, d_volP, voln, alpha, d_U, numP) != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! Device matrix multiplication error (U <- PX + H)\n");
		return EXIT_FAILURE;
	}


	// find max parallelly
	thrust::reduce_by_key(thrust::device, d_sampleId.begin(), d_sampleId.begin() + numP * voln, thrust::make_zip_iterator(thrust::make_tuple(d_cellId.begin(), d_U_ptr)), d_voln_key.begin(),
		thrust::make_zip_iterator(thrust::make_tuple(d_ind.begin(), d_ind_val.begin())), thrust::equal_to<int>(), find_max());

	// calculate histogram
	dense_histogram(d_ind, d_g);


	return 0;
}


int cuOMT_simple::gd_update_h()
{
	// subtract current cell measures with target ones
	thrust::transform(d_g.begin(), d_g.end(), d_A.begin(), d_g.begin(), axmy<float>(1.0f / voln));

	/*update h from g*/
#ifdef USE_FANCY_GD
    thrust::transform(thrust::device, d_adam_m_ptr, d_adam_m_ptr + numP, d_g.begin(), d_adam_m_ptr, update_adam_m<float>(0.9f));
    thrust::transform(thrust::device, d_adam_v_ptr, d_adam_v_ptr + numP, d_g.begin(), d_adam_v_ptr, update_adam_v<float>(0.999f));
    thrust::transform(thrust::device, d_adam_m_ptr, d_adam_m_ptr + numP, d_adam_v_ptr, d_delta_h.begin(), adam_delta_h(lr));
    thrust::transform(thrust::device, d_h.begin(), d_h.begin() + numP, d_delta_h.begin(), d_h.begin(), thrust::plus<float>());
#else
    thrust::transform(thrust::device, d_g.begin(), d_g.end(), d_delta_h.begin(), axpb<float>(-lr, 0.0f));
    thrust::transform(thrust::device, d_h.begin(), d_h.begin() + numP, d_delta_h.begin(), d_h.begin(), thrust::plus<float>());
#endif

	thrust::transform(d_h.begin(), d_h.end(), d_h.begin(), axpb<float>(1.0f, thrust::transform_reduce(d_h.begin(), d_h.end(), axpb<float>(-1.0f / (float)numP, 0.0f), 0.0f, thrust::plus<float>())));


	/*terminate condition*/
	d_g_norm[0] = sqrt(thrust::transform_reduce(d_g.begin(), d_g.end(), square<float>(), 0.0f, thrust::plus<float>()));
	//if (iter % 100 == 0)
	if (!quiet_mode)
		std::cout << "[" << iter << "/" << maxIter << "] g norm: " << d_g_norm[0] << "/" << eps << std::endl;
	if (d_g_norm[0] < d_eps[0])
		return 0;
	else ++iter;


	return 1;

}

void cuOMT_simple::write_pushed_mu(const char* output)
{
	// calculate histogram
	dense_histogram(d_ind, d_g);
	thrust::transform(d_g.begin(), d_g.end(), d_g.begin(), axpb<float>(1.0f / voln, 0));

	_get_to_csv(output, thrust::raw_pointer_cast(&d_g[0]), 1, numP);
}

void cuOMT_simple::write_h(const char* output)
{
	_get_to_csv(output, thrust::raw_pointer_cast(&d_h[0]), 1, numP);
}

void cuOMT_simple::write_volP(const char* output)
{
	_get_to_csv(output, thrust::raw_pointer_cast(&d_volP[0]), voln, dim);
}

void cuOMT_simple::write_ind(const char* output)
{
	_get_to_csv(output, thrust::raw_pointer_cast(&d_ind[0]), 1, voln);
}
#ifdef USE_FANCY_GD
void cuOMT_simple::write_adam_m(const char* output)
{
    _get_to_csv(output, d_adam_m, 1, numP);
}

void cuOMT_simple::write_adam_v(const char* output)
{
    _get_to_csv(output, d_adam_v, 1, numP);
}
#endif
void cuOMT_simple::write_generated_P(const char* output)
{
	_get_to_csv(output, d_P, numP, dim);
}

int cuOMT_simple::gd_clean(void)
{
	/* Memory clean up */

	if (hipFree(d_U) != hipSuccess)
	{
		fprintf(stderr, "!!!! memory free error (U)\n");
		return EXIT_FAILURE;
	}

	if (hipFree(d_volP) != hipSuccess)
	{
		fprintf(stderr, "!!!! memory free error (volP)\n");
		return EXIT_FAILURE;
	}

#ifdef USE_FANCY_GD
    if (hipFree(d_adam_m) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (adam_m)\n");
        return EXIT_FAILURE;
    }

    if (hipFree(d_adam_v) != hipSuccess)
    {
        fprintf(stderr, "!!!! memory free error (adam_v)\n");
        return EXIT_FAILURE;
    }
#endif

	/* shut down*/
	if (hipblasDestroy(handle) != HIPBLAS_STATUS_SUCCESS)
	{
		fprintf(stderr, "!!!! shutdown error (A)\n");
		return EXIT_FAILURE;
	}

	std::cout << "cuOMT finished successfully!\n(Press any key to continue...)" << std::endl;
	std::cin.get();
	return 0;

}

template <typename Vector1, typename Vector2>
void cuOMT_simple::dense_histogram(Vector1& input, Vector2& histogram)
{
	typedef typename Vector1::value_type ValueType; // input value type
	typedef typename Vector2::value_type IndexType; // histogram index type

													// copy input data (could be skipped if input is allowed to be modified)
													//thrust::device_vector<ValueType> data(input);

													// print the initial data
													//print_vector("initial data", data);

													// sort data to bring equal elements together
	thrust::device_vector<int> data(input);
	thrust::sort(data.begin(), data.end());

	// print the sorted data
	//print_vector("sorted data", input);

	// number of histogram bins is equal to the maximum value plus one
	int num_bins = data.back() + 1;

	// resize histogram storage
	//histogram.resize(num_bins);

	// find the end of each bin of values
	//thrust::counting_iterator<IndexType> search_begin(0);
	thrust::upper_bound(data.begin(), data.end(),
		thrust::make_counting_iterator<IndexType>(0), thrust::make_counting_iterator<IndexType>(num_bins),
		histogram.begin());

	// print the cumulative histogram
	//print_vector("cumulative histogram", histogram);

	// compute the histogram by taking differences of the cumulative histogram
	thrust::adjacent_difference(histogram.begin(), histogram.begin() + num_bins,
		histogram.begin());
	// fill outbounded historgram entry to 0
	if (num_bins < histogram.size())
		thrust::fill(histogram.begin() + num_bins, histogram.end(), 0.0f);

	// print the histogram
	//print_vector("histogram", histogram);
}

template <typename T>
void cuOMT_simple::print_matrix(T* A, int nr_rows_A, int nr_cols_A) {
	for (int i = 0; i < nr_rows_A; ++i) {
		for (int j = 0; j < nr_cols_A; ++j) {
			std::cout << A[j * nr_rows_A + i] << " ";

		}
		std::cout << std::endl;

	}
	std::cout << std::endl;
}

template <typename T>
void cuOMT_simple::print_matrix_csv(T* A, int nr_rows_A, int nr_cols_A, const char* output) {
	std::ofstream file;
	file.open(output);
	for (int i = 0; i < nr_rows_A; ++i) {
		for (int j = 0; j < nr_cols_A; ++j) {
			if (j != nr_cols_A - 1)
				file << std::setprecision(20) << A[j * nr_rows_A + i] << ",";
			else
				file << std::setprecision(20) << A[j * nr_rows_A + i] << "\n";
		}
	}
	file.close();
}

void cuOMT_simple::tic() {
	tictoc_stack.push(clock());
}

double cuOMT_simple::toc() {
	/*std::cout << "Time elapsed: "
	<< ((double)(clock() - tictoc_stack.top())) / CLOCKS_PER_SEC
	<< std::endl;*/
	auto t = ((double)(clock() - tictoc_stack.top())) / CLOCKS_PER_SEC;
	tictoc_stack.pop();
	return t;
}

template <typename T>
void cuOMT_simple::_get_to_csv(const char* output, T* d_vec, int row, int col)
{
	// compute centroid
	T* h_vec = 0;
	h_vec = (T *)malloc(row * col * sizeof(h_vec[0]));
	hipblasGetVector(row * col, sizeof(h_vec[0]), d_vec, 1, h_vec, 1);
	print_matrix_csv(h_vec, row, col, output);

	// free junk
	free(h_vec);
}

template <typename T>
void cuOMT_simple::_set_from_csv(const char* input, T* d_vec, int row, int col)
{

	thrust::host_vector<T> h_vec(row * col);
	std::ifstream file(input);
	int count = 0;
	std::string line, value;
	while (file.good())
	{
		getline(file, line);
		std::stringstream          lineStream(line);
		while (std::getline(lineStream, value, ','))
		{
			T val;
			if (typeid(T) == typeid(float))
				val = std::stof(value);
			else if (typeid(T) == typeid(int))
				val = std::stoi(value);
			h_vec[count] = val;
			++count;
		}
	}
	file.close();
	auto d_vec_ptr = thrust::device_pointer_cast(d_vec);
	thrust::copy(h_vec.begin(), h_vec.end(), d_vec_ptr);



	h_vec.clear();


}

template <typename T>
void cuOMT_simple::_set_random_parameter(T* d_para, const int row, const int col)
{
	GPU_generate_RNM(d_para, row, col);
}

void cuOMT_simple::run_simple_omt(int argc, char* argv[])
{
	gd_init(argc, argv);
	gd_pre_calc();
	tic();

	//output

	while (iter < maxIter && !gd_calc_measure() && gd_update_h())
	{
		if (!no_output)
		{
			std::string output_h = std::string("h/") + std::to_string(iter) + std::string(".csv");
            write_h(output_h.c_str());

            std::string output_mu = std::string("pushed_mu/") + std::to_string(iter) + std::string(".csv");
            write_pushed_mu(output_mu.c_str());


            std::string output_volP = std::string("volP/") + std::to_string(iter) + std::string(".csv");
            write_volP(output_volP.c_str());

            std::string output_ind = std::string("ind/") + std::to_string(iter) + std::string(".csv");
            write_ind(output_ind.c_str());
		}

	}
	std::cout << "cuOMT loop takes " << toc() << "s" << std::endl;
	write_h("./h/h.csv");
	write_volP("./volP/volP_final.csv");
	write_ind("./ind/ind_final.csv");
	gd_clean();
}

cuOMT_simple::cuOMT_simple(const int _dim, const int _numP, const int _voln, const int _maxIter, const float _eps, const float _lr, bool _no_output, bool _quiet_mode)
	:dim(_dim), numP(_numP), voln(_voln), maxIter(_maxIter), lr(_lr), eps(_eps), no_output(_no_output), quiet_mode(_quiet_mode)
{

}


